#include "hip/hip_runtime.h"
#include "optimizers.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "error_checking.h"
#include "dense_kernels.h"
#include <cstdio>

void adam_update(float* d_param, const float* d_grad,
                 float* d_m, float* d_v,
                 float learning_rate, float beta1, float beta2,
                 float epsilon, float globalIter, int size) {
    printf("adam_update: updating %d elements, lr=%f, globalIter=%f\n", size, learning_rate, globalIter);
    hipDeviceSynchronize();
}

void clip_gradients(float* d_arr, int n, float clip_val) {
    float* d_temp;
    CUDA_CHECK(hipMalloc(&d_temp, n * sizeof(float)));
    clipArray(d_arr, d_temp, n, clip_val);
    CUDA_CHECK(hipMemcpy(d_arr, d_temp, n * sizeof(float), hipMemcpyDeviceToDevice));
    hipFree(d_temp);
}

void clip_parameters(float* d_param, int size, float clip_val) {
    // Implement parameter clipping logic here
}

void clipGradientsCustom(hipblasHandle_t handle, float* d_grad, int size, float max_norm) {
    float norm = 0.0f;
    CUBLAS_CHECK(hipblasSnrm2(handle, size, d_grad, 1, &norm));
    if (norm > max_norm) {
        float scale = max_norm / norm;
        CUBLAS_CHECK(hipblasSscal(handle, size, &scale, d_grad, 1));
    }
}
