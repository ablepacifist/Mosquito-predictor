#include "hip/hip_runtime.h"
#include "conv_kernels.h"

// Definition of the convolution-specific bias addition kernel.
__global__ void addBiasKernelConv(float* output, const float* bias, int C, int N, int H, int W) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = N * C * H * W;
    if (idx < total) {
        int channel = (idx / (H * W)) % C;
        output[idx] += bias[channel];
    }
}
__global__ void clampKernel(float* data, int count, float minVal, float maxVal) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < count) {
        float val = data[idx];
        // Use the device function __isnanf for a robust check.
        if (__isnanf(val)) {
            data[idx] = 0.0f;
        } else {
            // Clamp with fminf and fmaxf.
            float clamped = fmaxf(minVal, fminf(val, maxVal));
            data[idx] = clamped;
        }
    }
}

__global__ void updateWeightsKernel(float *weights, const float *dW, float learning_rate, int num_elements)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elements)
    {
        weights[idx] -= learning_rate * dW[idx];
    }
}

