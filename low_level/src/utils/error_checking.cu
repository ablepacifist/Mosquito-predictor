#include "hip/hip_runtime.h"
#include <iostream>
#include <error_checking.h>
__global__ void checkForInvalidValues(float* array, int size, int* flag) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        if (isnan(array[idx]) || isinf(array[idx])) {
            atomicExch(flag, 1);
        }
    }
}

void verifyArray(float* d_array, int size, const char* msg) {
    int *d_flag;
    CUDA_CHECK(hipMalloc(&d_flag, sizeof(int)));
    CUDA_CHECK(hipMemset(d_flag, 0, sizeof(int)));

    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;
    checkForInvalidValues<<<gridSize, blockSize>>>(d_array, size, d_flag);
    hipDeviceSynchronize();

    int h_flag = 0;
    hipMemcpy(&h_flag, d_flag, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_flag);

    if (h_flag) {
        std::cerr << msg << " contains NaN or Inf!" << std::endl;
        exit(EXIT_FAILURE);
    }
}
