#include "global_average_pooling_layer.h"
#include <hipDNN.h>
#include <cstdlib>
#include <iostream>

void globalAveragePoolingForward(
    hipdnnHandle_t handle,
    const hipdnnTensorDescriptor_t inputDesc,
    const float *d_input,
    const hipdnnTensorDescriptor_t outputDesc,
    float *d_output)
{
    // Create a temporary pooling descriptor.
    hipdnnPoolingDescriptor_t gapDesc;
    hipdnnStatus_t status = hipdnnCreatePoolingDescriptor(&gapDesc);
    if (status != HIPDNN_STATUS_SUCCESS)
    {
        std::cerr << "Error creating pooling descriptor: "
                  << hipdnnGetErrorString(status) << std::endl;
        exit(EXIT_FAILURE);
    }

    // Retrieve spatial dimensions (H and W) from the input descriptor.
    hipdnnDataType_t dataType;
    int n, c, h, w;
    int nStride, cStride, hStride, wStride;

    status = hipdnnGetTensor4dDescriptor(
        inputDesc,
        &dataType,
        &n, &c, &h, &w,
        &nStride, &cStride, &hStride, &wStride
    );
    if (status != HIPDNN_STATUS_SUCCESS)
    {
        std::cerr << "Error retrieving tensor descriptor: "
                  << hipdnnGetErrorString(status) << std::endl;
        exit(EXIT_FAILURE);
    }

    // Debug: print the dimensions if needed.
    // std::cout << "Tensor dimensions: " << n << " " << c << " " << h << " " << w << "\n";

    // Set pooling window to cover the full spatial dimensions (global average pooling).
    status = hipdnnSetPooling2dDescriptor(
        gapDesc,
        HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING,
        HIPDNN_PROPAGATE_NAN,
        h, w,   // Use full height and width of the input feature map.
        0, 0,   // No padding.
        1, 1    // Stride 1.
    );
    if (status != HIPDNN_STATUS_SUCCESS)
    {
        std::cerr << "Error setting pooling descriptor: "
                  << hipdnnGetErrorString(status) << std::endl;
        exit(EXIT_FAILURE);
    }

    // Launch the pooling forward operation.
    float alpha = 1.0f, beta = 0.0f;
    status = hipdnnPoolingForward(
        handle,
        gapDesc,
        &alpha,
        inputDesc, d_input,
        &beta,
        outputDesc, d_output
    );
    if (status != HIPDNN_STATUS_SUCCESS)
    {
        std::cerr << "Error in global average pooling forward: "
                  << hipdnnGetErrorString(status) << std::endl;
        exit(EXIT_FAILURE);
    }

    // Clean up the pooling descriptor.
    hipdnnDestroyPoolingDescriptor(gapDesc);
}
