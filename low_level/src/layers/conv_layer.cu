#include "hip/hip_runtime.h"
#include "../include/layers/conv_layer.h"
#include "../include/utils/error_checking.h"
#include "../include/utils/conv_kernels.h"
#include "../include/utils/dense_kernels.h"
#include "../include/utils/weight_init.h"
#include "../include/optimizers/optimizers.h"
#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cstdlib>

// Helper: Returns the total number of elements from a cudnn tensor descriptor.
int getTotalElements(hipdnnTensorDescriptor_t tensorDesc) {
    int n, c, h, w, s_n, s_c, s_h, s_w;
    hipdnnDataType_t dataType;
    CUDNN_CHECK(hipdnnGetTensor4dDescriptor(tensorDesc, &dataType, &n, &c, &h, &w,
                                           &s_n, &s_c, &s_h, &s_w));
    return n * c * h * w;
}

// ConvLayer Implementation (Forward Pass Only)
ConvLayer::ConvLayer(hipdnnHandle_t cudnn,
                     int in_channels,
                     int out_channels,
                     int kernelH, int kernelW,
                     int padH, int padW,
                     int strideH, int strideW,
                     int dilationH, int dilationW)
    : cudnn(cudnn), in_channels(in_channels), out_channels(out_channels),
      kernelH(kernelH), kernelW(kernelW), padH(padH), padW(padW),
      strideH(strideH), strideW(strideW), dilationH(dilationH), dilationW(dilationW),
      d_filter(nullptr), d_bias(nullptr), d_output(nullptr),
      d_filter_m(nullptr), d_filter_v(nullptr)
{
    int filterSize = out_channels * in_channels * kernelH * kernelW;
    CUDA_CHECK(hipMalloc(&d_filter, filterSize * sizeof(float)));
    float stddev = sqrtf(2.0f / static_cast<float>(in_channels * kernelH * kernelW));
    initializeWeights(d_filter, filterSize, stddev);

    CUDA_CHECK(hipMalloc(&d_bias, out_channels * sizeof(float)));
    CUDA_CHECK(hipMemset(d_bias, 0, out_channels * sizeof(float)));
}

ConvLayer::~ConvLayer() {
    if (d_filter) hipFree(d_filter);
    if (d_bias) hipFree(d_bias);
    if (d_output) hipFree(d_output);
    if (d_filter_m) hipFree(d_filter_m);
    if (d_filter_v) hipFree(d_filter_v);
}

int ConvLayer::forward(hipdnnTensorDescriptor_t inputDesc, float* d_input) {
    // Get input dimensions
    int n, c, h, w;
    int nStride, cStride, hStride, wStride;
    hipdnnDataType_t dt;
    CUDNN_CHECK(hipdnnGetTensor4dDescriptor(inputDesc, &dt, &n, &c, &h, &w,
                                           &nStride, &cStride, &hStride, &wStride));
    int inputElements = n * c * h * w;

    // Clamp input and fix NaN/Inf
    int blockSize = 256;
    int gridSize = (inputElements + blockSize - 1) / blockSize;
    float minClamp = -1.0f, maxClamp = 1.0f;
    clampKernel<<<gridSize, blockSize>>>(d_input, inputElements, minClamp, maxClamp);
    hipDeviceSynchronize();
    checkCudaError("clampKernel in ConvLayer::forward()");
    fixNaNInfKernel<<<gridSize, blockSize>>>(d_input, inputElements);
    hipDeviceSynchronize();

    // Create convolution descriptor
    hipdnnConvolutionDescriptor_t convDesc;
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&convDesc));
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(
        convDesc, padH, padW, strideH, strideW, dilationH, dilationW,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    // Create filter descriptor
    hipdnnFilterDescriptor_t filterDesc;
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&filterDesc));
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(
        filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
        out_channels, in_channels, kernelH, kernelW));

    // Determine output dimensions
    int N_out, C_out, H_out, W_out;
    CUDNN_CHECK(hipdnnGetConvolution2dForwardOutputDim(
        convDesc, inputDesc, filterDesc, &N_out, &C_out, &H_out, &W_out));
    hipdnnTensorDescriptor_t outputDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&outputDesc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(
        outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N_out, C_out, H_out, W_out));

    // Allocate output buffer
    size_t outputSize = static_cast<size_t>(N_out) * C_out * H_out * W_out;
    if (d_output) hipFree(d_output);
    CUDA_CHECK(hipMalloc(&d_output, outputSize * sizeof(float)));

    // Choose convolution algorithm and allocate workspace
    hipdnnConvolutionFwdAlgo_t fwdAlgo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
    size_t workspaceSize = 0;
    CUDNN_CHECK(hipdnnGetConvolutionForwardWorkspaceSize(
        cudnn, inputDesc, filterDesc, convDesc, outputDesc, fwdAlgo, &workspaceSize));
    void* d_workspace = nullptr;
    if (workspaceSize > 0) {
        CUDA_CHECK(hipMalloc(&d_workspace, workspaceSize));
    }

    // Perform convolution
    float alpha = 1.0f, beta = 0.0f;
    CUDNN_CHECK(hipdnnConvolutionForward(
        cudnn, &alpha, inputDesc, d_input, filterDesc, d_filter,
        convDesc, fwdAlgo, d_workspace, workspaceSize, &beta, outputDesc, d_output));
    if (d_workspace) hipFree(d_workspace);
    hipDeviceSynchronize();
    checkCudaError("hipdnnConvolutionForward in ConvLayer::forward()");

    // Fix NaN/Inf in convolution output
    gridSize = (outputSize + blockSize - 1) / blockSize;
    fixNaNInfKernel<<<gridSize, blockSize>>>(d_output, outputSize);
    hipDeviceSynchronize();
    if (containsNaNorInf(d_output, outputSize)) {
        std::cerr << "NaN/Inf detected after convolution in ConvLayer::forward()" << std::endl;
        exit(EXIT_FAILURE);
    }

    // Add bias
    int totalElements = N_out * C_out * H_out * W_out;
    int blockSizeBias = 256;
    int gridSizeBias = (totalElements + blockSizeBias - 1) / blockSizeBias;
    addBiasKernelConv<<<gridSizeBias, blockSizeBias>>>(d_output, d_bias, C_out, N_out, H_out, W_out);
    CUDA_CHECK(hipDeviceSynchronize());

    // Fix NaN/Inf after bias addition
    fixNaNInfKernel<<<gridSize, blockSize>>>(d_output, outputSize);
    hipDeviceSynchronize();
    if (containsNaNorInf(d_output, outputSize)) {
        std::cerr << "NaN/Inf detected after bias addition in ConvLayer::forward()" << std::endl;
        exit(EXIT_FAILURE);
    }

    // Apply ReLU activation
    hipdnnActivationDescriptor_t actDesc;
    CUDNN_CHECK(hipdnnCreateActivationDescriptor(&actDesc));
    CUDNN_CHECK(hipdnnSetActivationDescriptor(actDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.0));
    CUDNN_CHECK(hipdnnActivationForward(cudnn, actDesc, &alpha, outputDesc, d_output, &beta, outputDesc, d_output));
    CUDNN_CHECK(hipdnnDestroyActivationDescriptor(actDesc));
    hipDeviceSynchronize();
    checkCudaError("hipdnnActivationForward in ConvLayer::forward()");

    // Final NaN/Inf check after activation
    fixNaNInfKernel<<<gridSize, blockSize>>>(d_output, outputSize);
    hipDeviceSynchronize();
    if (containsNaNorInf(d_output, outputSize)) {
        std::cerr << "NaN/Inf detected after activation in ConvLayer::forward()" << std::endl;
        exit(EXIT_FAILURE);
    }

    // Clean up descriptors
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(outputDesc));
    CUDNN_CHECK(hipdnnDestroyFilterDescriptor(filterDesc));
    CUDNN_CHECK(hipdnnDestroyConvolutionDescriptor(convDesc));

    // Return flattened output dimension
    return C_out * H_out * W_out;
}

// Backward pass: computes filter gradients and updates weights
void ConvLayer::backward(hipdnnTensorDescriptor_t inputDesc, float *d_input, float *d_output_grad_const)
{
    // Create convolution descriptor
    hipdnnConvolutionDescriptor_t convDesc;
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&convDesc));
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(
        convDesc, padH, padW, strideH, strideW, dilationH, dilationW,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    // Create filter descriptor
    hipdnnFilterDescriptor_t filterDesc;
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&filterDesc));
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(
        filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
        out_channels, in_channels, kernelH, kernelW));

    int filterElements = out_channels * in_channels * kernelH * kernelW;

    // Allocate memory for filter gradients
    float *d_filter_grad;
    CUDA_CHECK(hipMalloc(&d_filter_grad, filterElements * sizeof(float)));

    // Create output tensor descriptor
    hipdnnTensorDescriptor_t outputDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&outputDesc));
    int out_n, out_c, out_h, out_w;
    CUDNN_CHECK(hipdnnGetConvolution2dForwardOutputDim(convDesc, inputDesc, filterDesc,
                                                       &out_n, &out_c, &out_h, &out_w));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(
        outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, out_n, out_c, out_h, out_w));

    // Choose backward filter algorithm and allocate workspace
    hipdnnConvolutionBwdFilterAlgo_t algo = HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0;
    size_t workspaceSize = 0;
    CUDNN_CHECK(hipdnnGetConvolutionBackwardFilterWorkspaceSize(
        cudnn, inputDesc, outputDesc, convDesc, filterDesc, algo, &workspaceSize));
    void* d_workspace = nullptr;
    if (workspaceSize > 0) {
        CUDA_CHECK(hipMalloc(&d_workspace, workspaceSize));
    }

    // Compute filter gradients
    float alpha = 1.0f, beta = 0.0f;
    CUDNN_CHECK(hipdnnConvolutionBackwardFilter(
        cudnn, &alpha,
        inputDesc, d_input,
        outputDesc, d_output_grad_const,
        convDesc, algo,
        d_workspace, workspaceSize,
        &beta,
        filterDesc, d_filter_grad));
    if (d_workspace)
        hipFree(d_workspace);
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(outputDesc));

    // Fix NaN/Inf in gradients
    int blockSize = 256;
    int gridSize = (filterElements + blockSize - 1) / blockSize;
    fixNaNInfKernel<<<gridSize, blockSize>>>(d_filter_grad, filterElements);
    hipDeviceSynchronize();

    // Update filter weights
    float learning_rate = 0.001f;
    updateWeightsKernel<<<gridSize, blockSize>>>(d_filter, d_filter_grad, learning_rate, filterElements);
    CUDA_CHECK(hipDeviceSynchronize());

    hipFree(d_filter_grad);

    // Clean up descriptors
    CUDNN_CHECK(hipdnnDestroyConvolutionDescriptor(convDesc));
    CUDNN_CHECK(hipdnnDestroyFilterDescriptor(filterDesc));
}

float* ConvLayer::getOutput() const {
    return d_output;
}
