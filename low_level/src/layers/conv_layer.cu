#include "hip/hip_runtime.h"
#include "../include/layers/conv_layer.h"
#include "../include/utils/error_checking.h"
#include "../include/utils/conv_kernels.h"     // Provides addBiasKernelConv, leakyReluKernel, leakyReluDerivativeKernel.
#include "../include/utils/dense_kernels.h"   // Provides addBiasKernelConv, leakyReluKernel, leakyReluDerivativeKernel.
#include "../include/utils/weight_init.h"     // Provides initializeWeights.
#include "../include/optimizers/optimizers.h" // Provides adam_update and clip_gradients.
#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cstdlib>           // For rand()


// Constructor: allocate and initialize filter weights and biases.
// Helper: Returns the total number of elements from a cudnn tensor descriptor.
int getTotalElements(hipdnnTensorDescriptor_t tensorDesc) {
    int n, c, h, w, s_n, s_c, s_h, s_w;
    hipdnnDataType_t dataType;
    CUDNN_CHECK(hipdnnGetTensor4dDescriptor(tensorDesc, &dataType, &n, &c, &h, &w,
                                           &s_n, &s_c, &s_h, &s_w));
    return n * c * h * w;
}

/////////////////////////////////////////////////////////////////
// ConvLayer Implementation (Forward Pass Only)
/////////////////////////////////////////////////////////////////

ConvLayer::ConvLayer(hipdnnHandle_t cudnn,
                     int in_channels,
                     int out_channels,
                     int kernelH, int kernelW,
                     int padH, int padW,
                     int strideH, int strideW,
                     int dilationH, int dilationW)
    : cudnn(cudnn), in_channels(in_channels), out_channels(out_channels),
      kernelH(kernelH), kernelW(kernelW), padH(padH), padW(padW),
      strideH(strideH), strideW(strideW), dilationH(dilationH), dilationW(dilationW),
      d_filter(nullptr), d_bias(nullptr), d_output(nullptr),
      d_filter_m(nullptr), d_filter_v(nullptr)
{
    // Compute the total number of filter elements.
    int filterSize = out_channels * in_channels * kernelH * kernelW;
    CUDA_CHECK(hipMalloc(&d_filter, filterSize * sizeof(float)));
    // Initialize filters using your weight initializer (He initialization).
    float stddev = sqrtf(2.0f / static_cast<float>(in_channels * kernelH * kernelW));
    initializeWeights(d_filter, filterSize, stddev);
    
    // Allocate bias vector (one per output channel).
    CUDA_CHECK(hipMalloc(&d_bias, out_channels * sizeof(float)));
    CUDA_CHECK(hipMemset(d_bias, 0, out_channels * sizeof(float)));
}

ConvLayer::~ConvLayer() {
    if (d_filter) hipFree(d_filter);
    if (d_bias) hipFree(d_bias);
    if (d_output) hipFree(d_output);
    if (d_filter_m) hipFree(d_filter_m);
    if (d_filter_v) hipFree(d_filter_v);
}

int ConvLayer::forward(hipdnnTensorDescriptor_t inputDesc, float* d_input) {
    // 1. Get input dimensions.
    int n, c, h, w;
    int nStride, cStride, hStride, wStride;
    hipdnnDataType_t dt;  
    CUDNN_CHECK(hipdnnGetTensor4dDescriptor(inputDesc, &dt, &n, &c, &h, &w,
                                           &nStride, &cStride, &hStride, &wStride));
    int inputElements = n * c * h * w;

    // 2. Clamp d_input to the range [-1, 1] and fix NaN values.
    int blockSize = 256;
    int gridSize = (inputElements + blockSize - 1) / blockSize;
    float minClamp = -1.0f, maxClamp = 1.0f;
    clampKernel<<<gridSize, blockSize>>>(d_input, inputElements, minClamp, maxClamp);
    hipDeviceSynchronize();
    checkCudaError("clampKernel in ConvLayer::forward()");
    // Apply a fix kernel to replace any NaN/Inf in the input.
    fixNaNInfKernel<<<gridSize, blockSize>>>(d_input, inputElements);
    hipDeviceSynchronize();

    // 3. Create convolution descriptor.
    hipdnnConvolutionDescriptor_t convDesc;
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&convDesc));
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(
        convDesc, padH, padW, strideH, strideW, dilationH, dilationW,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    // 4. Create filter descriptor.
    hipdnnFilterDescriptor_t filterDesc;
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&filterDesc));
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(
        filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
        out_channels, in_channels, kernelH, kernelW));

    // 5. Determine output dimensions.
    int N_out, C_out, H_out, W_out;
    CUDNN_CHECK(hipdnnGetConvolution2dForwardOutputDim(
        convDesc, inputDesc, filterDesc, &N_out, &C_out, &H_out, &W_out));
    hipdnnTensorDescriptor_t outputDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&outputDesc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(
        outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N_out, C_out, H_out, W_out));

    // 6. Allocate output buffer.
    size_t outputSize = static_cast<size_t>(N_out) * C_out * H_out * W_out;
    if (d_output) hipFree(d_output);
    CUDA_CHECK(hipMalloc(&d_output, outputSize * sizeof(float)));

    // 7. Choose convolution algorithm and allocate workspace.
    hipdnnConvolutionFwdAlgo_t fwdAlgo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
    size_t workspaceSize = 0;
    CUDNN_CHECK(hipdnnGetConvolutionForwardWorkspaceSize(
        cudnn, inputDesc, filterDesc, convDesc, outputDesc, fwdAlgo, &workspaceSize));
    void* d_workspace = nullptr;
    if (workspaceSize > 0) {
        CUDA_CHECK(hipMalloc(&d_workspace, workspaceSize));
    }

    // 8. Perform convolution.
    float alpha = 1.0f, beta = 0.0f;
    CUDNN_CHECK(hipdnnConvolutionForward(
        cudnn, &alpha, inputDesc, d_input, filterDesc, d_filter,
        convDesc, fwdAlgo, d_workspace, workspaceSize, &beta, outputDesc, d_output));
    if (d_workspace) hipFree(d_workspace);
    hipDeviceSynchronize();
    checkCudaError("hipdnnConvolutionForward in ConvLayer::forward()");

    // 9. Fix any NaN/Inf in the convolution output.
    gridSize = (outputSize + blockSize - 1) / blockSize;
    fixNaNInfKernel<<<gridSize, blockSize>>>(d_output, outputSize);
    hipDeviceSynchronize();
    if (containsNaNorInf(d_output, outputSize)) {
        std::cerr << "NaN/Inf detected after convolution in ConvLayer::forward()" << std::endl;
        exit(EXIT_FAILURE);
    }

    // 10. Add bias via kernel.
    int totalElements = N_out * C_out * H_out * W_out;
    int blockSizeBias = 256;
    int gridSizeBias = (totalElements + blockSizeBias - 1) / blockSizeBias;
    addBiasKernelConv<<<gridSizeBias, blockSizeBias>>>(d_output, d_bias, C_out, N_out, H_out, W_out);
    CUDA_CHECK(hipDeviceSynchronize());

    // 11. Fix any NaN/Inf after bias addition.
    fixNaNInfKernel<<<gridSize, blockSize>>>(d_output, outputSize);
    hipDeviceSynchronize();
    if (containsNaNorInf(d_output, outputSize)) {
        std::cerr << "NaN/Inf detected after bias addition in ConvLayer::forward()" << std::endl;
        exit(EXIT_FAILURE);
    }

    // 12. Apply ReLU activation via cuDNN.
    hipdnnActivationDescriptor_t actDesc;
    CUDNN_CHECK(hipdnnCreateActivationDescriptor(&actDesc));
    CUDNN_CHECK(hipdnnSetActivationDescriptor(actDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.0));
    CUDNN_CHECK(hipdnnActivationForward(cudnn, actDesc, &alpha, outputDesc, d_output, &beta, outputDesc, d_output));
    CUDNN_CHECK(hipdnnDestroyActivationDescriptor(actDesc));
    hipDeviceSynchronize();
    checkCudaError("hipdnnActivationForward in ConvLayer::forward()");

    // 13. Final check after activation.
    fixNaNInfKernel<<<gridSize, blockSize>>>(d_output, outputSize);
    hipDeviceSynchronize();
    if (containsNaNorInf(d_output, outputSize)) {
        std::cerr << "NaN/Inf detected after activation in ConvLayer::forward()" << std::endl;
        exit(EXIT_FAILURE);
    }

    // 14. Clean up descriptors.
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(outputDesc));
    CUDNN_CHECK(hipdnnDestroyFilterDescriptor(filterDesc));
    CUDNN_CHECK(hipdnnDestroyConvolutionDescriptor(convDesc));

    // Return the flattened output dimension (i.e. C_out * H_out * W_out).
    return C_out * H_out * W_out;
}






void ConvLayer::backward(hipdnnTensorDescriptor_t inputDesc, float *d_input, float *d_output_grad_const)
{
    // 1. Create convolution descriptor.
    hipdnnConvolutionDescriptor_t convDesc;
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&convDesc));
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(
        convDesc, padH, padW, strideH, strideW, dilationH, dilationW,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    // 2. Create filter descriptor.
    hipdnnFilterDescriptor_t filterDesc;
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&filterDesc));
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(
        filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
        out_channels, in_channels, kernelH, kernelW));

    // 3. Compute total number of filter elements.
    int filterElements = out_channels * in_channels * kernelH * kernelW;

    // 4. Allocate memory for filter gradients.
    float *d_filter_grad;
    CUDA_CHECK(hipMalloc(&d_filter_grad, filterElements * sizeof(float)));

    // 5. Create output tensor descriptor (for convolution forward output).
    hipdnnTensorDescriptor_t outputDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&outputDesc));
    int out_n, out_c, out_h, out_w;
    CUDNN_CHECK(hipdnnGetConvolution2dForwardOutputDim(convDesc, inputDesc, filterDesc,
                                                       &out_n, &out_c, &out_h, &out_w));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(
        outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, out_n, out_c, out_h, out_w));

    // 6. Choose a backward filter algorithm.
    hipdnnConvolutionBwdFilterAlgo_t algo = HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0;

    // 7. Query workspace size.
    size_t workspaceSize = 0;
    CUDNN_CHECK(hipdnnGetConvolutionBackwardFilterWorkspaceSize(
        cudnn, inputDesc, outputDesc, convDesc, filterDesc, algo, &workspaceSize));
    void* d_workspace = nullptr;
    if (workspaceSize > 0) {
        CUDA_CHECK(hipMalloc(&d_workspace, workspaceSize));
    }

    // 8. Compute the gradient with respect to filter weights.
    float alpha = 1.0f, beta = 0.0f;
    CUDNN_CHECK(hipdnnConvolutionBackwardFilter(
        cudnn, &alpha,
        inputDesc, d_input,
        outputDesc, d_output_grad_const,
        convDesc, algo,
        d_workspace, workspaceSize,
        &beta,
        filterDesc, d_filter_grad));
    if (d_workspace)
        hipFree(d_workspace);
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(outputDesc));

    // 9. Fix any NaN/Inf found in the computed gradients.
    int blockSize = 256;
    int gridSize = (filterElements + blockSize - 1) / blockSize;
    fixNaNInfKernel<<<gridSize, blockSize>>>(d_filter_grad, filterElements);
    hipDeviceSynchronize();

    // 10. Update the filter weights using the computed gradients.
    float learning_rate = 0.001f;
    updateWeightsKernel<<<gridSize, blockSize>>>(d_filter, d_filter_grad, learning_rate, filterElements);
    CUDA_CHECK(hipDeviceSynchronize());

    // 11. Free the temporary gradient memory.
    hipFree(d_filter_grad);

    // 12. Destroy descriptors.
    CUDNN_CHECK(hipdnnDestroyConvolutionDescriptor(convDesc));
    CUDNN_CHECK(hipdnnDestroyFilterDescriptor(filterDesc));
}









float* ConvLayer::getOutput() const {
    return d_output;
}
