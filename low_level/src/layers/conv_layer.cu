#include "hip/hip_runtime.h"
#include "../include/layers/conv_layer.h"
#include "../include/utils/error_checking.h"
#include "../include/utils/conv_kernels.h"     // Provides addBiasKernelConv, leakyReluKernel, leakyReluDerivativeKernel.
#include "../include/utils/dense_kernels.h"   // Provides addBiasKernelConv, leakyReluKernel, leakyReluDerivativeKernel.
#include "../include/utils/weight_init.h"     // Provides initializeWeights.
#include "../include/optimizers/optimizers.h" // Provides adam_update and clip_gradients.
#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cstdlib>           // For rand()


// Constructor: allocate and initialize filter weights and biases.
// Helper: Returns the total number of elements from a cudnn tensor descriptor.
int getTotalElements(hipdnnTensorDescriptor_t tensorDesc) {
    int n, c, h, w, s_n, s_c, s_h, s_w;
    hipdnnDataType_t dataType;
    CUDNN_CHECK(hipdnnGetTensor4dDescriptor(tensorDesc, &dataType, &n, &c, &h, &w,
                                           &s_n, &s_c, &s_h, &s_w));
    return n * c * h * w;
}

/////////////////////////////////////////////////////////////////
// ConvLayer Implementation (Forward Pass Only)
/////////////////////////////////////////////////////////////////

ConvLayer::ConvLayer(hipdnnHandle_t cudnn,
                     int in_channels,
                     int out_channels,
                     int kernelH, int kernelW,
                     int padH, int padW,
                     int strideH, int strideW,
                     int dilationH, int dilationW)
    : cudnn(cudnn), in_channels(in_channels), out_channels(out_channels),
      kernelH(kernelH), kernelW(kernelW), padH(padH), padW(padW),
      strideH(strideH), strideW(strideW), dilationH(dilationH), dilationW(dilationW),
      d_filter(nullptr), d_bias(nullptr), d_output(nullptr),
      d_filter_m(nullptr), d_filter_v(nullptr)
{
    // Compute the total number of filter elements.
    int filterSize = out_channels * in_channels * kernelH * kernelW;
    CUDA_CHECK(hipMalloc(&d_filter, filterSize * sizeof(float)));
    // Initialize filters using your weight initializer (He initialization).
    float stddev = sqrtf(2.0f / static_cast<float>(in_channels * kernelH * kernelW));
    initializeWeights(d_filter, filterSize, stddev);
    
    // Allocate bias vector (one per output channel).
    CUDA_CHECK(hipMalloc(&d_bias, out_channels * sizeof(float)));
    CUDA_CHECK(hipMemset(d_bias, 0, out_channels * sizeof(float)));
}

ConvLayer::~ConvLayer() {
    if (d_filter) hipFree(d_filter);
    if (d_bias) hipFree(d_bias);
    if (d_output) hipFree(d_output);
    if (d_filter_m) hipFree(d_filter_m);
    if (d_filter_v) hipFree(d_filter_v);
}

int ConvLayer::forward(hipdnnTensorDescriptor_t inputDesc, float* d_input) {
    // Get input dimensions from the tensor descriptor.
    int n, c, h, w;
    int nStride, cStride, hStride, wStride;
    hipdnnDataType_t dt;  // Declare a variable for the data type.
    CUDNN_CHECK(hipdnnGetTensor4dDescriptor(inputDesc, &dt, &n, &c, &h, &w,
                                           &nStride, &cStride, &hStride, &wStride));
    int inputElements = n * c * h * w;
    
    //std::cout << "Input dimensions: N=" << n << ", C=" << c << ", H=" << h << ", W=" << w << std::endl;
    //std::cout << "Total input elements: " << inputElements << std::endl;
    
    // Clamp d_input: force values into the range [-1.0f, 1.0f] and replace any NaN with 0.0f.
    {
        int blockSize = 256;
        int gridSize = (inputElements + blockSize - 1) / blockSize;
        float minClamp = -1.0f;
        float maxClamp = 1.0f;
        clampKernel<<<gridSize, blockSize>>>(d_input, inputElements, minClamp, maxClamp);
        hipDeviceSynchronize();
        checkCudaError("clampKernel in forward()");
    }
    
    // Log statistics for d_input after clamping using printArrayStats from error_checking.h.
    //printArrayStats(d_input, inputElements, "Input Tensor (d_input) after clamping");
    
    // Create a convolution descriptor.
    hipdnnConvolutionDescriptor_t convDesc;
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&convDesc));
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(convDesc,
                                                padH, padW,
                                                strideH, strideW,
                                                dilationH, dilationW,
                                                HIPDNN_CROSS_CORRELATION,
                                                HIPDNN_DATA_FLOAT));
    
    // Create a filter descriptor.
    hipdnnFilterDescriptor_t filterDesc;
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&filterDesc));
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(filterDesc,
                                           HIPDNN_DATA_FLOAT,
                                           HIPDNN_TENSOR_NCHW,
                                           out_channels,
                                           in_channels,
                                           kernelH, kernelW));
    
    // Get output dimensions using cuDNN.
    int N_out, C_out, H_out, W_out;
    CUDNN_CHECK(hipdnnGetConvolution2dForwardOutputDim(convDesc, inputDesc, filterDesc, &N_out, &C_out, &H_out, &W_out));
    
    // Create an output tensor descriptor.
    hipdnnTensorDescriptor_t outputDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&outputDesc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N_out, C_out, H_out, W_out));
    
    // Allocate output buffer.
    size_t outputSize = static_cast<size_t>(N_out) * C_out * H_out * W_out;
    if (d_output) hipFree(d_output);
    CUDA_CHECK(hipMalloc(&d_output, outputSize * sizeof(float)));
    
    // Choose a convolution forward algorithm.
    hipdnnConvolutionFwdAlgo_t fwdAlgo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
    
    // Get workspace size.
    size_t workspaceSize = 0;
    CUDNN_CHECK(hipdnnGetConvolutionForwardWorkspaceSize(cudnn, inputDesc, filterDesc, convDesc, outputDesc, fwdAlgo, &workspaceSize));
    void* d_workspace = nullptr;
    if (workspaceSize > 0) {
        CUDA_CHECK(hipMalloc(&d_workspace, workspaceSize));
    }
    
    float alpha = 1.0f, beta = 0.0f;
    // Perform the convolution.
    CUDNN_CHECK(hipdnnConvolutionForward(cudnn,
                                          &alpha,
                                          inputDesc,
                                          d_input,
                                          filterDesc,
                                          d_filter,
                                          convDesc,
                                          fwdAlgo,
                                          d_workspace,
                                          workspaceSize,
                                          &beta,
                                          outputDesc,
                                          d_output));
    if (d_workspace) hipFree(d_workspace);
    
    hipDeviceSynchronize();
    checkCudaError("hipdnnConvolutionForward in ConvLayer::forward()");
    
    // Check for NaN/Inf immediately after convolution.
    if (containsNaNorInf(d_output, outputSize)) {
        std::cerr << "NaN/Inf detected after convolution in ConvLayer::forward()" << std::endl;
        exit(EXIT_FAILURE);
    }
    
    // Add bias using a kernel.
    int totalElements = N_out * C_out * H_out * W_out;
    {
        int blockSizeBias = 256;
        int gridSizeBias = (totalElements + blockSizeBias - 1) / blockSizeBias;
        addBiasKernelConv<<<gridSizeBias, blockSizeBias>>>(d_output, d_bias, C_out, N_out, H_out, W_out);
        CUDA_CHECK(hipDeviceSynchronize());
    }
    
    if (containsNaNorInf(d_output, outputSize)) {
        std::cerr << "NaN/Inf detected after bias addition in ConvLayer::forward()" << std::endl;
        exit(EXIT_FAILURE);
    }
    
    // Apply ReLU activation via cuDNN.
    hipdnnActivationDescriptor_t actDesc;
    CUDNN_CHECK(hipdnnCreateActivationDescriptor(&actDesc));
    CUDNN_CHECK(hipdnnSetActivationDescriptor(actDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.0));
    CUDNN_CHECK(hipdnnActivationForward(cudnn, actDesc, &alpha, outputDesc, d_output, &beta, outputDesc, d_output));
    CUDNN_CHECK(hipdnnDestroyActivationDescriptor(actDesc));
    
    hipDeviceSynchronize();
    checkCudaError("hipdnnActivationForward in ConvLayer::forward()");
    
    if (containsNaNorInf(d_output, outputSize)) {
        std::cerr << "NaN/Inf detected after activation in ConvLayer::forward()" << std::endl;
        exit(EXIT_FAILURE);
    }
    
    // Clean up descriptors.
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(outputDesc));
    CUDNN_CHECK(hipdnnDestroyFilterDescriptor(filterDesc));
    CUDNN_CHECK(hipdnnDestroyConvolutionDescriptor(convDesc));
    
    // Return the flattened output dimension (C_out * H_out * W_out).
    return C_out * H_out * W_out;
}






void ConvLayer::backward(hipdnnTensorDescriptor_t inputDesc, float *d_input, float *d_output_grad_const)
{
    // Create a convolution descriptor using the layer parameters.
    hipdnnConvolutionDescriptor_t convDesc;
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&convDesc));
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(convDesc,
                                                padH, padW,         // padding
                                                strideH, strideW,   // strides
                                                dilationH, dilationW, // dilation
                                                HIPDNN_CROSS_CORRELATION,
                                                HIPDNN_DATA_FLOAT));

    // Create a filter descriptor for our filter weights.
    hipdnnFilterDescriptor_t filterDesc;
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&filterDesc));
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(filterDesc,
                                           HIPDNN_DATA_FLOAT,
                                           HIPDNN_TENSOR_NCHW,
                                           out_channels, in_channels, kernelH, kernelW));

    // Compute total number of filter elements.
    int filterElements = out_channels * in_channels * kernelH * kernelW;

    // Allocate device memory for the computed filter gradients.
    float *d_filter_grad;
    CUDA_CHECK(hipMalloc(&d_filter_grad, filterElements * sizeof(float)));

    // Create an output tensor descriptor to represent the convolution output.
    hipdnnTensorDescriptor_t outputDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&outputDesc));
    int out_n, out_c, out_h, out_w;
    CUDNN_CHECK(hipdnnGetConvolution2dForwardOutputDim(convDesc, inputDesc, filterDesc,
                                                       &out_n, &out_c, &out_h, &out_w));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(outputDesc,
                                           HIPDNN_TENSOR_NCHW,
                                           HIPDNN_DATA_FLOAT,
                                           out_n, out_c, out_h, out_w));

    // Choose a backward filter algorithm.
    // Using ALGO_0 as a fallback if the more advanced algorithm is unavailable.
    hipdnnConvolutionBwdFilterAlgo_t algo = HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0;

    // Query the workspace size for this backward filter computation.
    size_t workspaceSize = 0;
    CUDNN_CHECK(hipdnnGetConvolutionBackwardFilterWorkspaceSize(
                  cudnn,
                  inputDesc,
                  outputDesc,
                  convDesc,
                  filterDesc,
                  algo,
                  &workspaceSize));
    void* d_workspace = nullptr;
    if (workspaceSize > 0)
    {
        CUDA_CHECK(hipMalloc(&d_workspace, workspaceSize));
    }

    // Compute the gradient with respect to the filter weights.
    float alpha = 1.0f, beta = 0.0f;
    CUDNN_CHECK(hipdnnConvolutionBackwardFilter(cudnn,
                  &alpha,
                  inputDesc, d_input,
                  outputDesc, d_output_grad_const,
                  convDesc,
                  algo,
                  d_workspace, workspaceSize,
                  &beta,
                  filterDesc, d_filter_grad));

    // Free workspace memory if allocated.
    if (d_workspace)
        hipFree(d_workspace);
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(outputDesc));

    // Update the filter weights using the computed gradients.
    float learning_rate = 0.001f;  // Set your desired learning rate.
    const int blockSize = 256;
    int gridSize = (filterElements + blockSize - 1) / blockSize;
    updateWeightsKernel<<<gridSize, blockSize>>>(d_filter, d_filter_grad, learning_rate, filterElements);
    CUDA_CHECK(hipDeviceSynchronize());

    // Free the temporary gradient memory.
    hipFree(d_filter_grad);

    // Destroy the descriptors we created.
    CUDNN_CHECK(hipdnnDestroyConvolutionDescriptor(convDesc));
    CUDNN_CHECK(hipdnnDestroyFilterDescriptor(filterDesc));
}








float* ConvLayer::getOutput() const {
    return d_output;
}
